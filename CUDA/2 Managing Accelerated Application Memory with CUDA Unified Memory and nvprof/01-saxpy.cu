#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nvprof to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * a, int * b, int * c)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = tid; i < N; i += stride)
  {
        c[i] = 2 * a[i] + b[i];
  }
}

__global__ void init(int * a, int * b, int * c)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = tid; i < N; i += stride)
  {
    a[i] = 2;
    b[i] = 1;
    c[i] = 0;
  }
}


int main()
{
  int deviceId;
  hipGetDevice(&deviceId);
  
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;

    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

   hipMemPrefetchAsync(a, size, deviceId);
   hipMemPrefetchAsync(b, size, deviceId);
   hipMemPrefetchAsync(c, size, deviceId);

    // Initialize memory
    /*
    for( int i = 0; i < N; ++i )
    {
        a[i] = 2;
        b[i] = 1;
        c[i] = 0;
    }
    */

    //int threads_per_block = 128;
    //int number_of_blocks = (N / threads_per_block) + 1;
    int number_of_blocks = multiProcessorCount;
    int threads_per_block = warpSize * 32;


    init <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    //hipDeviceSynchronize();
    
    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    
    hipError_t syncErr;
    hipError_t asyncErr;
    
    syncErr = hipGetLastError();
    if(syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));

    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    hipMemPrefetchAsync(c, size, hipCpuDeviceId);

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    
    hipFree( a ); hipFree( b ); hipFree( c );
}
