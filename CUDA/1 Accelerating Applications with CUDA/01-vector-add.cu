
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}


void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  /*
  for(int i = 0; i < N; ++i)
  {
    result[i] = a[i] + b[i];
  }
  */
  int gridStride = gridDim.x * blockDim.x;
  for( int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += gridStride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  //a = (float *)malloc(size);
  //b = (float *)malloc(size);
  //c = (float *)malloc(size);
  hipMallocManaged( &a, size );
  hipMallocManaged( &b, size );
  hipMallocManaged( &c, size );


  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  int blockNum = 32;
  int tNum = 1024;
  addVectorsInto<<<blockNum, tNum>>>(c, a, b, N);
  checkCuda( hipDeviceSynchronize() );  // Check sync errors
  checkCuda( hipGetLastError() );       // Check async errors

  checkElementsAre(7, c, N);

  //free(a);
  //free(b);
  //free(c);
  hipFree( a );
  hipFree( b );
  hipFree( c );
}
