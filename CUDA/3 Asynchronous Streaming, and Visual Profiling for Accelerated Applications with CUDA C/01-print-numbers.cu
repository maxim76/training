
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printNumber(int number)
{
  printf("%d\n", number);
}

int main()
{
  hipStream_t streams[5];
  for (int i = 0; i < 5; ++i) {
      hipStreamCreate(&(streams[i]));
  }
  
  for (int i = 0; i < 5; ++i)
  {
    printNumber<<<1, 1, 0, streams[i]>>>(i);
  }
  hipDeviceSynchronize();
  
  for (int i = 0; i < 5; ++i) {
      hipStreamDestroy(streams[i]);
  }
  
}
