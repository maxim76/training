
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;
  float *h_c;

  hipMalloc(&a, size);
  hipMalloc(&b, size);
  hipMalloc(&c, size);
  hipHostMalloc(&h_c, size, hipHostMallocDefault);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  /*
   * Create 3 streams to run initialize the 3 data vectors in parallel.
   */

  hipStream_t stream1, stream2, stream3;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);

  /*
   * Give each `initWith` launch its own non-standard stream.
   */

  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, c, N);



  const int numberOfSegments = 4;                  // This example demonstrates slicing the work into 4 segments.
  int segmentN = N / numberOfSegments;             // A value for a segment's worth of `N` is needed.
  size_t segmentSize = size / numberOfSegments;    // A value for a segment's worth of `size` is needed.

  // For each of the 4 segments...
  for (int i = 0; i < numberOfSegments; ++i)
  {
    // Calculate the index where this particular segment should operate within the larger arrays.
    size_t segmentOffset = i * segmentN;

    // Create a stream for this segment's worth of copy and work.
    hipStream_t stream;
    hipStreamCreate(&stream);
  

    addVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream>>>(&c[segmentOffset], &a[segmentOffset], &b[segmentOffset], segmentN);

    //cudaMemcpy(h_c, c, size, cudaMemcpyDeviceToHost);
    hipMemcpyAsync(&h_c[segmentOffset], &c[segmentOffset], segmentSize, hipMemcpyHostToDevice, stream);  
                  
    addVectorsErr = hipGetLastError();
    if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));
    hipStreamDestroy(stream);
    
  }

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, h_c, N);

  /*
   * Destroy streams when they are no longer needed.
   */

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);

  hipFree(a);
  hipFree(b);
  hipFree(c);
  hipHostFree(h_c);
}
